#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
/*
Tested blocks/threads: 
256/256
256/1024
128/1024
512/1024
*/
#define NUM_BLOCK  512  
#define NUM_THREAD  1024  
#define PI  3.14159265358979323846  

/* Kernel function */
__global__ void cal_pi(double *mypi, int iter, 
    double m, int nthreads, int nblocks) {
	int i;
	double ni;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    // Sequential thread index across the blocks
	for (i = idx; i< iter; i += nthreads * nblocks) {
		ni = (i + 0.5) * m;
		mypi[idx] += 4.0/(1.0 + ni * ni);
	}
}


int main(void) {
    double pi = 0;
	clock_t start,end;
    int iteArr[3] = { 24000000, 48000000, 94000000 };
    //double iteArr[3] = { 24000000000, 48000000000, 94000000000 };
    //int iteArr[3] = { 1, 2, 4 };

    /* Setting upt grid and block dimesions */
    dim3 dimGrid(NUM_BLOCK,1,1);  
    dim3 dimBlock(NUM_THREAD,1,1); 
    printf("REPORT # of blocks = %d, # of threads/block = %d\n", NUM_BLOCK, NUM_THREAD);

    /* Host and Device variables (arrays) */
    double *h_pi, *d_pi;  
    for (int i = 0; i < 3; i++){
        int currentIter = iteArr[i];

        start = clock(); // because why not...
        double step = 1.0 / currentIter;  
        size_t size = NUM_BLOCK*NUM_THREAD*sizeof(double);  
        
        /* Allocate on host*/
        h_pi = (double *)malloc(size);  
    
        /*Allocate on device*/
        hipMalloc((void **) &d_pi, size);  
        /* Set d_pi to zero */
        hipMemset(d_pi, 0, size);
        /* Run Kernel */
        cal_pi <<<dimGrid, dimBlock>>> (d_pi, currentIter, 
            step, NUM_THREAD, NUM_BLOCK); 
    
        /* Copy results from device to the host*/
        hipMemcpy(h_pi, d_pi, size, hipMemcpyDeviceToHost);
        
        /* Finish pi in host */
        for( int j = 0; j < NUM_THREAD*NUM_BLOCK; j++)
            pi += h_pi[j];
        pi *= step;
        printf("\tMyPI = %20.18f \n",pi);
        printf("\tMyPI - PI = %20.18f \n",pi-PI);
        end = clock();
 
        /* Print Report */
        printf("\tTime estimation in CPU : %f s.\n\n",(end-start)/(float)CLOCKS_PER_SEC);
    }

    printf("\tCheck nvprof for more time estimation.\n");
    	
    /* Clean host and device var*/
	free(h_pi); 
	hipFree(d_pi);

	return 0;
}