#include <stdio.h>
#include <hip/hip_runtime.h>



#define NUM_BLOCK 256
#define NUM_THREAD 256
#define PI 3.14159265358979323846264

__global__
void calcPI(double *mypi, int iter, double m, int nthreads,
    int nblocks){
        
        double ni;
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        for(int i = index; i < iter; i += nthreads*nblocks){
            ni = ((double)i + 0.5) * m;
            *mypi += 4.0 / (1.0 + ni * ni);
        }
        
}

int main (void) {
    printf("START\n");
    dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions
    dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions
    int iteArr[3] = { 24000000, 48000000, 94000000 };
    double size, m, *h_pi, *d_pi;
    size = sizeof(double);

    h_pi = (double*)malloc(size);
    hipMalloc((void **) &d_pi, size);
/*
    for(int i = 0; i < 3; i++){
        m = 1.0 / iteArr[i];
        cudaMemset(d_pi, 0, size);
        calcPI<<<dimGrid, dimBlock>>>(d_pi,iteArr[i],m,NUM_THREAD,NUM_BLOCK);
        cudaMemcpy(h_pi,d_pi, size, cudaMemcpyDeviceToHost);
        h_pi *= m;
    }
*/
        int i = 0;
        m = 1.0 / (double)iteArr[i];
        hipMemset(d_pi, 0, size);

        printf("iteArr = %d\n", iteArr[i]);
        printf("m = %d\n", m);
        printf("Threads = %d\n", NUM_THREAD);
        printf("Blocks = %d\n", NUM_BLOCK);

        calcPI<<<dimGrid, dimBlock>>>(d_pi,iteArr[i],m,NUM_THREAD,NUM_BLOCK);
        hipMemcpy(h_pi,d_pi, size, hipMemcpyDeviceToHost);
        printf("before mypi: %f\n", *h_pi);
        *h_pi *= m;
        printf("after mypi: %f\n", *h_pi);

    free(h_pi);
    hipFree(d_pi);
    return 0;
}