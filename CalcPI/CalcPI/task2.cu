#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define PI 3.14159265358979323846264
__global__ void calcPi(int *iter, double *result){
	double m, ni, diff, mypi = 0.0; 
	m = 1.0/(double)*iter;

	for(int i = 0; i < *iter; i++){
		ni =((double)i + 0.5) * m;
		mypi += 4.0 / (1.0 + ni * ni);
	}

	mypi *= m;
	diff = mypi - PI;
	*result = diff;
}


int main (void) {
	printf("START\n");
	int arrLength = 3;
	int iteArr[3] = { 240, 480, 960 };
	double myResults[3];

	int *d_ite, ite;
	double *d_result, h_result;
	int sizeInt = sizeof(int);
	double sizeDou = sizeof(double);

/*
 * Allocate space for device copies
 */
	hipMalloc((void**) &d_ite, sizeInt);
	hipMalloc((void**) &d_result, sizeDou);

for(int j = 0; j < arrLength; j++){
	printf("Running %d j: %d\n", iteArr[j], j);
	ite = iteArr[j];
/*
* Copy input
*/
	hipMemcpy(d_ite, &ite, sizeInt, hipMemcpyHostToDevice);
	//cudaMemcpy(d_result, &result, sizeDou , cudaMemcpyHostToDevice);
/*
* Launch Kernel
*/
	calcPi<<<1,1>>>(d_ite, d_result);
/*
* Copy result back to host
*/
	hipMemcpy(&h_result, d_result, sizeDou, hipMemcpyDeviceToHost);
	//printf("result %.70f\n", h_result);

    myResults[j] = h_result;
    
	//printf("mypi %f\n", mypi[j]);
	
/*
* Clean up
*/
hipFree(d_ite); hipFree(d_result);


}//for loop

for(int k = 0; k < 3; k++){

	printf("RESULT %d =  %.70f\n", k, myResults[k]);
}
printf("PASSED\n");

return 0;


}
