#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define PI 3.14159265358979323846264
__global__ void calcPi(int *iter, double *result){
	double m, ni, diff, mypi = 0.0; 
	m = 1.0/(double)*iter;

	for(int i = 0; i < *iter; i++){
		ni =((double)i + 0.5) * m;
		mypi += 4.0 / (1.0 + ni * ni);
	}

	mypi *= m;
	diff = mypi - PI;
	*result = diff;
}


int main (void) {
	printf("START\n");
	int arrLength = 3;
	int iteArr[3] = { 24000000, 48000000, 94000000 };
	double myResults[3];

	int *d_ite, ite;
	double *d_result, h_result;
	int sizeInt = sizeof(int);
	double sizeDou = sizeof(double);
	

/*
 * Allocate space for device copies. 
 * @params (void**) pointer, size
 */
	hipMalloc((void**) &d_ite, sizeInt);
	hipMalloc((void**) &d_result, sizeDou);

	float time;
	hipEvent_t start, stop;
	
for(int j = 0; j < arrLength; j++){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	printf("Running %d j: %d\n", iteArr[j], j);
	ite = iteArr[j];
/*
* Copy input
* @params (pointer dst, pointer src, size, type)
*/
	hipMemcpy(d_ite, &ite, sizeInt, hipMemcpyHostToDevice);
	//cudaMemcpy(d_result, &result, sizeDou , cudaMemcpyHostToDevice);
/*
* Launch Kernel
*/
	calcPi<<<1,1>>>(d_ite, d_result);
/*
* Copy result back to host
* @params (pointer dst, pointer src, size, type)
*/
	hipMemcpy(&h_result, d_result, sizeDou, hipMemcpyDeviceToHost);
	//printf("result %.70f\n", h_result);

    myResults[j] = h_result;
    
	//printf("mypi %f\n", mypi[j]);
		
	/*
	* Clean up
	*/	
	hipFree(d_ite); hipFree(d_result);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time /= 1000;
	printf("Time to generate:  %f s \n", time);




}//for loop

for(int k = 0; k < 3; k++){

	printf("RESULT %d =  %.70f\n", k, myResults[k]);
}
printf("PASSED\n");

return 0;


}
/*
* Notes: cuda uses pointers. 
*/