#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#include <iostream>
#include <math.h>
#define PI 3.14159265358979323846264
// Kernel function to add the elements of two arrays
__global__
void calcPI(int iter, double *m, double *ni, double *mypi)
{
    /*
    *m = 1.0/(double) iter;

    for(int i = 0; i < iter; i++){
        *ni = ((double) i + 0.5) * *m;
        *mypi += 4.0 / (1.0 + *ni * *ni);
    }
    *mypi *= *m;  
    */
    int index = threadIdx.x;
    int stride = blockDim.x;
    *m = 1.0/(double) iter;

    for(int i = index; i < iter; i+= stride){
        *ni = ((double) i + 0.5) * *m;
        *mypi  += 4.0 / (1.0 + *ni * *ni);
    }
    *mypi  *= *m; 
}

int main(void)
{
    printf("===TEST N 19===\n");
    
    int N;
    double *m;
    double *ni;
    double *mypi;
    int iteArr[3] = { 24000000, 48000000, 94000000 };
    //int iteArr[3] = { 100,200,300};

    double results[3] = { 0, 0, 0 };

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&m, sizeof(double));
    hipMallocManaged(&ni, sizeof(double));
    hipMallocManaged(&mypi, sizeof(double));
    
    for(int i = 0; i < 3; i++){
        // initialize variables on the host
        *m = 0.0;
        *ni = 0.0;
        *mypi = 0.0;
    
        N = iteArr[i];
        
        // Run kernel on  the GPU
        calcPI<<<1, 1>>>(N, m, ni, mypi);

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();
        
        
        // Check for errors (all values should be 3.0f)
        results[i] = *mypi;
        printf("Result in loop: %d %f \n", N, *mypi);
    }
    
    for (int j = 0; j < 3; j++){
        printf("\n- Result array: %f\n", results[j]);
    }


    // Free memory
    hipFree(m);
    hipFree(ni);
    hipFree(mypi);
    return 0;
}